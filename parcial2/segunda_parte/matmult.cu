#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "matmult.h"
#define N 100
#define tile 32

//Multiplicacion secuencial
void sec_matMult(int* A, int aCol, int aRow, int* B, int bCol, int bRow, int* C){
	for (int i = 0; i < aRow; i++) {
    for (int j = 0; j < bCol; j++) {
   		int sum = 0;
      for (int k = 0; k < aCol; k++) {
        sum += A[j * aCol + k] * B[k * aCol + i];
      }
   	  C[j * aCol + i] = sum;
  	}
 	}
}

//Multiplicacion memoria global
__global__ void gbmem_matMult(int* m1, int* m2, int* ans, int n){
	int k, sum = 0;
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < n && j < n) {
    for (k = 0; k < n; k++) {
      sum += m1[j * n + k] * m2[k * n + i];
    }
    ans[j * n + i] = sum;
  }
}

//Multiplicacion memoria compartida
__global__ void sdmem_matMult(int* m1, int* m2, int* ans, int n){

  __shared__ int m1_s[tile][tile];
  __shared__ int m2_s[tile][tile];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int row = by * tile + ty;
  int col = bx * tile + tx;

	int sum = 0;
	for(int m = 0; m < n/tile; ++m){
    m1_s[ty][tx] = m1[row * n + m * tile + tx];
    m2_s[ty][tx] = m2[(m * tile + ty) * n + col];
    __syncthreads();

    for (int k = 0; k < tile; ++k) {
      sum += m1_s[ty][k] * m2_s[k][tx];
    }
    __syncthreads();
  }
  ans[row * n + col] = sum;
}


int main(int argc, char** argv ){

  //Definicion de variables
  FILE *f1, *f2, *f3, *f4, *f5;
  double secTime, globalTime, sharedTime;
  int *h_m1, *h_m2, *h_ans;
  int *d_m1, *d_m2, *d_ans;
  int m1Row, m1Col, m2Row, m2Col; 

  if (argc != 2){
    printf("Cantidad de parametros incorrecta!!\n");
  }else{
    //Creación de archivos
    matGen(atoi(argv[1]));
    //f1 = fopen(argv[1],"r");
    //f2 = fopen(argv[2],"r");
    f1 = fopen("mat1.txt","r");
    f2 = fopen("mat2.txt","r");
    f3 = fopen("sec_ans.txt","w");
    f4 = fopen("glo_ans.txt","w");
    f5 = fopen("sha_ans.txt","w");
    //Lectura de dimensiones de las matrices
    fscanf(f1, "%d", &m1Row); fscanf(f1, "%d", &m1Col);
    fscanf(f2, "%d", &m2Row); fscanf(f2, "%d", &m2Col);

    //Definición de tamaño para asignar memoria
    size_t m1Size = m1Row * m1Col * sizeof(int);
    size_t m2Size = m2Row * m2Col * sizeof(int);
    size_t ansSize = m1Col * m2Row * sizeof(int);

    //Asignación de memoria en el Host
    h_m1 = (int *)malloc(m1Size);
    h_m2 = (int *)malloc(m2Size);
    h_ans = (int *)malloc(ansSize);

    //Lectura de archivos y almacenamiento en el Host
    readAllocFile(f1, h_m1, m1Row, m1Col);
    readAllocFile(f2, h_m2, m2Row, m2Col);

    //Asignacion de memoria en el Device
    printf("- Asignacion de memoria en el Device... ");
    if (hipSuccess != hipMalloc((void **) &d_m1, m1Size))
      printf("Error asignando para d_m1\n");
    if (hipSuccess != hipMalloc((void **) &d_m2, m2Size))
      printf("Error asignando para d_m2\n");
    if (hipSuccess != hipMalloc((void **) &d_ans, ansSize))
      printf("Error asignando para d_ans\n");
    printf("OK!!!\n");

    //Copia de datos del Host al Device
    printf("- Copia de datos H -> D... ");
    if (hipSuccess != hipMemcpy(d_m1, h_m1, m1Size, hipMemcpyHostToDevice))
      printf("Error copiando d_m1\n");
	  if (hipSuccess != hipMemcpy(d_m2, h_m2, m2Size, hipMemcpyHostToDevice))
      printf("Error copiando d_m2\n");
    printf("OK!!!\n");

    printf("Tiempo:\n");

    //Llamado a la multiplicacion secuencial
    clock_t startSecTime = clock();
    sec_matMult(h_m1, m1Col, m1Row, h_m2, m2Col, m2Row, h_ans);
    secTime = ((double)(clock()-startSecTime))/CLOCKS_PER_SEC;
    printf("> Secuencial = %.6fs\n",secTime);

    //Generacion de archivo respuesta
    //setAnsFile("secuencial", m1Row, m2Col, h_ans, f3);

    /////////////////////////////////////

    int threads = m1Row;//Cantidad de hilos

    //Definicion de estructuras para cantidad de Hilos y Bloques
    dim3 blockDim(tile,tile);
	  dim3 gridDim((int)ceil((float)threads/blockDim.x), (int)ceil((float)threads/blockDim.y));

    //Multiplicacion paralela con memoria global
    clock_t startGlobalTime = clock();
    //Llamado al Kernel
    gbmem_matMult<<<gridDim, blockDim>>>(d_m1, d_m2, d_ans, threads);
    if(hipSuccess != hipGetLastError())
      printf("Error en el llamado al kernel (gMem)\n");

    //Copia de datos del Device al Host
    if (hipSuccess != hipMemcpy(h_ans, d_ans, ansSize, hipMemcpyDeviceToHost))
      printf("Error copiando datos desde d_ans a h_ans (gMem)\n");
    globalTime = ((double)(clock()-startGlobalTime))/CLOCKS_PER_SEC;
    printf("> Memoria global (cuda) = %.6fs\n",globalTime);

    //setAnsFile("global-mem", m1Row, m2Col, h_ans, f4);

    ///////////////////////////////////////

    //Multiplicacion paralela con memoria compartida
    clock_t startSharedTime = clock();
    //Llamado al Kernel
    sdmem_matMult<<<gridDim, blockDim>>>(d_m1, d_m2, d_ans, threads);
    if(hipSuccess != hipGetLastError())
      printf("Error en el llamado al kernel (sMem)\n");

    //Copia de datos del Device al Host
    if (hipSuccess != hipMemcpy(h_ans, d_ans, ansSize, hipMemcpyDeviceToHost))
      printf("Error copiando datos desde d_ans a h_ans (sMem)\n");
    sharedTime = ((double)(clock()-startSharedTime))/CLOCKS_PER_SEC;
    printf("> Memoria compartida (cuda) = %.6fs\n",sharedTime);

    //setAnsFile("shared-mem", m1Row, m2Col, h_ans, f5);

    //Liberacion de memoria
    free(h_m1); free(h_m2); free(h_ans);
	  hipFree(d_m1); hipFree(d_m2); hipFree(d_ans);

  }

}
