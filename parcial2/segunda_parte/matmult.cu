#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "matmult.h"
#define N 100
#define tile 32

//Multiplicacion secuencial
void sec_matMult(int* A, int aCol, int aRow, int* B, int bCol, int bRow, int* C){
	for (int i = 0; i < aRow; i++) {
    for (int j = 0; j < bCol; j++) {
   		int sum = 0;
      for (int k = 0; k < aCol; k++) {
        sum += A[j * aCol + k] * B[k * aCol + i];
      }
   	  C[j * aCol + i] = sum;
  	}
 	}
}

//Multiplicacion memoria global
__global__ void gbmem_matMult(int* m1, int* m2, int* ansG, int n){
	int k, sum = 0;
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < n && j < n) {
    for (k = 0; k < n; k++) {
      sum += m1[j * n + k] * m2[k * n + i];
    }
    ansG[j * n + i] = sum;
  }
}

//Multiplicacion memoria compartida
__global__ void sdmem_matMult(int* m1, int* m2, int* ansS, int n){

  __shared__ int m1_s[tile][tile];
  __shared__ int m2_s[tile][tile];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int row = by * tile + ty;
  int col = bx * tile + tx;

	int sum = 0;
	for(int m = 0; m < n/tile; ++m){
    m1_s[ty][tx] = m1[row * n + m * tile + tx];
    m2_s[ty][tx] = m2[(m * tile + ty) * n + col];
    __syncthreads();

    for (int k = 0; k < tile; ++k) {
      sum += m1_s[ty][k] * m2_s[k][tx];
    }
    __syncthreads();
  }
  ansS[row * n + col] = sum;
}


int main(int argc, char** argv ){

  //Definicion de variables
  FILE *f1, *f2, *f3, *f4, *f5;
  double secTime, globalTime, sharedTime;
  int *h_m1, *h_m2, *h_ans, *h_ansG, *h_ansS;
  int *d_m1, *d_m2, *d_ansG, *d_ansS;
  int m1Row, m1Col, m2Row, m2Col; 

  hipError_t err = hipSuccess;

  if (argc != 2){
    printf("Cantidad de parametros incorrecta!!\n");
  }else{
    //Creación de archivos
    matGen(atoi(argv[1]));
    //f1 = fopen(argv[1],"r");
    //f2 = fopen(argv[2],"r");
    f1 = fopen("mat1.txt","r");
    f2 = fopen("mat2.txt","r");
    f3 = fopen("sec_ans.txt","w");
    f4 = fopen("glo_ans.txt","w");
    f5 = fopen("sha_ans.txt","w");
    //Lectura de dimensiones de las matrices
    fscanf(f1, "%d", &m1Row); fscanf(f1, "%d", &m1Col);
    fscanf(f2, "%d", &m2Row); fscanf(f2, "%d", &m2Col);

    //Definición de tamaño para asignar memoria
    size_t m1Size = m1Row * m1Col * sizeof(int);
    size_t m2Size = m2Row * m2Col * sizeof(int);
    size_t ansSize = m1Col * m2Row * sizeof(int);

    //Asignación de memoria en el Host
    h_m1 = (int *)malloc(m1Size);
    h_m2 = (int *)malloc(m2Size);
    h_ans = (int *)malloc(ansSize);
    h_ansG = (int *)malloc(ansSize);
    h_ansS = (int *)malloc(ansSize);

    //Lectura de archivos y almacenamiento en el Host
    readAllocFile(f1, h_m1, m1Row, m1Col);
    readAllocFile(f2, h_m2, m2Row, m2Col);

    //Asignacion de memoria en el Device
    printf("> Asignacion de memoria en el Device...\n");
    err = hipMalloc((void **) &d_m1, m1Size);
    if(err != hipSuccess) printf(" -hipMalloc d_m1: %s\n",hipGetErrorString(err));
    err = hipMalloc((void **) &d_m2, m2Size);
    if(err != hipSuccess) printf(" -hipMalloc d_m2: %s\n",hipGetErrorString(err));
    err = hipMalloc((void **) &d_ansG, ansSize);
    if(err != hipSuccess) printf(" -hipMalloc d_ansG: %s\n",hipGetErrorString(err));
    err = hipMalloc((void **) &d_ansS, ansSize);
    if(err != hipSuccess) printf(" -hipMalloc d_ansS: %s\n",hipGetErrorString(err));

    //Copia de datos del Host al Device
    printf("> Copia de datos H -> D...\n");
    printf(" -hipMemcpy h_m1 -> d_m1: %s\n",hipGetErrorString(hipMemcpy(d_m1, h_m1, m1Size, hipMemcpyHostToDevice)));
    printf(" -hipMemcpy h_m2 -> d_m2: %s\n",hipGetErrorString(hipMemcpy(d_m2, h_m2, m1Size, hipMemcpyHostToDevice)));

    printf("Tiempo de ejecucion:\n");

    //Llamado a la multiplicacion secuencial
    clock_t startSecTime = clock();
    sec_matMult(h_m1, m1Col, m1Row, h_m2, m2Col, m2Row, h_ans);
    secTime = ((double)(clock()-startSecTime))/CLOCKS_PER_SEC;
    printf("> Secuencial = %.6fs\n",secTime);

    //Generacion de archivo respuesta
    //setAnsFile("secuencial", m1Row, m2Col, h_ans, f3);

    /////////////////////////////////////

    int threads = m1Row;//Cantidad de hilos

    //Definicion de estructuras para cantidad de Hilos y Bloques
    dim3 blockDim(tile,tile);
	  dim3 gridDim((int)ceil((float)threads/blockDim.x), (int)ceil((float)threads/blockDim.y));

    //Multiplicacion paralela con memoria global
    clock_t startGlobalTime = clock();
    gbmem_matMult<<<gridDim, blockDim>>>(d_m1, d_m2, d_ansG, threads);
    if(hipSuccess != hipGetLastError())
      printf("Error en el llamado al kernel (global-mem)\n");

    //Copia de datos del Device al Host
    if (hipSuccess != hipMemcpy(h_ansG, d_ansG, ansSize, hipMemcpyDeviceToHost))
      printf("Error copiando datos desde d_ansG a h_ansG (global-mem)\n");
    globalTime = ((double)(clock()-startGlobalTime))/CLOCKS_PER_SEC;
    printf("> Memoria global (cuda) = %.6fs\n",globalTime);
    hipDeviceSynchronize();

    //setAnsFile("global-mem", m1Row, m2Col, h_ansG, f4);

    ///////////////////////////////////////

    //Multiplicacion paralela con memoria compartida
    clock_t startSharedTime = clock();
    sdmem_matMult<<<gridDim, blockDim>>>(d_m1, d_m2, d_ansS, threads);
    if(hipSuccess != hipGetLastError())
      printf("Error en el llamado al kernel (shared-mem)\n");

    //Copia de datos del Device al Host
    hipError_t e = hipMemcpy(h_ansS, d_ansS, ansSize, hipMemcpyDeviceToHost);
    if (hipSuccess != e)
      printf("Error copiando datos desde d_ansS a h_ansS (shared-mem)\n (%s)\n",hipGetErrorString(e));
    sharedTime = ((double)(clock()-startSharedTime))/CLOCKS_PER_SEC;
    printf("> Memoria compartida (cuda) = %.6fs\n",sharedTime);

    //setAnsFile("shared-mem", m1Row, m2Col, h_ansS, f5);

    //Liberacion de memoria
    free(h_m1); free(h_m2); free(h_ans);
	  hipFree(d_m1); hipFree(d_m2); hipFree(d_ansG); hipFree(d_ansS);

  }

}
