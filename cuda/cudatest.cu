
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define N 4

__global__ void gpuMatmult(int* m1, int* m2, int* ans, int n){
	int k, sum = 0;
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < n && j < n) {
    for (k = 0; k < n; k++) {
      sum += m1[j * n + k] * m2[k * n + i];
    }
    ans[j * n + i] = sum;
  }
}

int main(){

	size_t bytes = N * N * sizeof(int);

	int *h_m1, *h_m2, *h_ans, *d_m1, *d_m2, *d_ans;

	h_m1 = (int *)malloc(bytes);
	h_m2 = (int *)malloc(bytes);
	h_ans = (int *)malloc(bytes);

	for(int i = 0;i < N * N ;i++){
		h_m1[i] = i;
		h_m2[i] = i;
		h_ans[i] = 0;
	}

	hipMalloc((void **) &d_m1, bytes);
	hipMalloc((void **) &d_m2, bytes);
	hipMalloc((void **) &d_ans, bytes);

	hipMemcpy(d_m1, h_m1, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_m2, h_m2, bytes, hipMemcpyHostToDevice);

	dim3 blockDim(32,32);
	dim3 gridDim((int)ceil((float)N/blockDim.x), (int)ceil((float)N/blockDim.y));

	gpuMatmult<<<gridDim, blockDim>>>(d_m1, d_m2, d_ans, N);

	hipMemcpy(h_ans, d_ans, bytes, hipMemcpyDeviceToHost);

	printf("%d\n",h_ans[1]);

	free(h_m1); free(h_m2); free(h_ans);
	hipFree(d_m1); hipFree(d_m2); hipFree(h_ans);

}