#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

using namespace cv;
using namespace std;

void imgToDec(unsigned char *imgBin, unsigned char *imgDec, int cols, int rows){//Cols must be cols x 3
    int pixelByChannel = 0;
    for(int row = 0; row < rows; row++){
        for(int col = 0; col < cols; col++){
            pixelByChannel = 0;
            for(int i = 7; i >= 0; i--){
                if(imgBin[(row * cols + col) * 8 + i] == 1) pixelByChannel += pow(2,7-i);
                // printf("%d\n",pixelByChannel);
            }
            imgDec[row * cols + col] = pixelByChannel;
        }   
    }
}

void imgToBin(unsigned char *imgDec, unsigned char *imgBin, int cols, int rows){//Cols must be cols x 3
    int pixelByChannel = 0;
    for(int row = 0; row < rows; row++){
        for(int col = 0; col < cols; col++){
            pixelByChannel = imgDec[row * cols + col];
            for(int i = 7; i >= 0; i--){
                imgBin[(row * cols + col) * 8 + i] = pixelByChannel%2;
                pixelByChannel = (pixelByChannel/2);
            }
        }   
    }
}


void hideImage(unsigned char *secImg, unsigned char *covImg, unsigned char *steImg, int cols, int rows){
    int secBit, covBit;
    for(int row = 0; row < rows; row++){
        for(int col = 0; col < cols; col++){
            for(int i = 7; i >= 4; i--){
                secBit = secImg[(row * cols + col) * 8 + (i-4)];
                covBit = covImg[(row * cols + col) * 8 + (i-4)];
                steImg[(row * cols + col) * 8 + (i-4)] = covBit;
                steImg[(row * cols + col) * 8 + i] = secBit;
            }
        }   
    }
}

void getSecImg(unsigned char *steImg, unsigned char *secImg, int cols, int rows){
    int secBit;
    for(int row = 0; row < rows; row++){
        for(int col = 0; col < cols; col++){
            for(int i = 7; i >= 4; i--){
                secBit = steImg[(row * cols + col) * 8 + i];
                secImg[(row * cols + col) * 8 + (i-4)] = secBit;
            }
        }   
    }
}

__global__ void imgToBinGPU(unsigned char *imgDec, unsigned char *imgBin, int cols, int rows){//Cols must be cols x 3
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int pixelByChannel = 0;
    if(row < rows && col < (cols * 8){
        pixelByChannel = imgDec[row * cols + col];
        for(int i = 7; i >= 0; i--){
            imgBin[(row * cols + col) * 8 + i] = pixelByChannel % 2;
            // printf("%d",pixelByChannel % 2);
            pixelByChannel = (pixelByChannel / 2);
        } 
    }
}

int main(int argc, char** argv )
{
    unsigned char *h_secImgRGB, *h_secImgBin, *h_secImgRec;
    unsigned char *h_covImgRGB, *h_covImgBin; 
    unsigned char *h_steImgRGB, *h_steImgBin;

    unsigned char *d_secImgRGB, *d_secImgBin, *d_secImgRec;
    unsigned char *d_covImgRGB, *d_covImgBin; 
    unsigned char *d_steImgRGB, *d_steImgBin;

    hipError_t err = hipSuccess;

    double timeCPU, timeGPU;
    
    if ( argc != 3 )
    {
        printf("usage: DisplayImage. <secretImg_Path> <coverImg_Path\n");
        return -1;
    }

    Mat secretImg, coverImg, stegoImg, recovImg;

    secretImg = imread(argv[1], 1);
    coverImg = imread(argv[2], 1);

    if ( !secretImg.data )
    {
        printf("No secretImg data \n");
        return -1;
    }

    printf("cov > %d x %d\nsec > %d x %d\n",coverImg.rows, coverImg.cols, secretImg.rows, secretImg.cols);

    int rows = secretImg.rows;
    int cols = secretImg.cols;
    int colsRGB = cols * secretImg.channels();
    int colsRGB_bin = cols * secretImg.channels() * 8;

    int imgSize = sizeof(unsigned char) * cols * rows * secretImg.channels();
    int imgSizeBin = sizeof(unsigned char) * cols * rows * secretImg.channels() * 8;

    h_secImgRGB = (unsigned char*)malloc(imgSize);
    h_secImgBin = (unsigned char*)malloc(imgSizeBin);
    h_secImgRec = (unsigned char*)malloc(imgSize);

    h_covImgRGB = (unsigned char*)malloc(imgSize);
    h_covImgBin = (unsigned char*)malloc(imgSizeBin);

    h_steImgRGB = (unsigned char*)malloc(imgSize);
    h_steImgBin = (unsigned char*)malloc(imgSizeBin);

    err = hipMalloc((void**)&d_secImgRGB, imgSize);
    if(err != hipSuccess){ printf(" -hipMalloc d_secImgRGB: %s\n",hipGetErrorString(err)); return 0;}

    err = hipMalloc((void**)&d_secImgBin, imgSizeBin);
    if(err != hipSuccess){ printf(" -hipMalloc d_secImgBin: %s\n",hipGetErrorString(err)); return 0;}

    err = hipMalloc((void**)&d_secImgRec, imgSize);
    if(err != hipSuccess){ printf(" -hipMalloc d_secImgRec: %s\n",hipGetErrorString(err)); return 0;}

    err = hipMalloc((void**)&d_covImgRGB, imgSize);
    if(err != hipSuccess){ printf(" -hipMalloc d_covImgRGB: %s\n",hipGetErrorString(err)); return 0;}

    err = hipMalloc((void**)&d_covImgBin, imgSizeBin);
    if(err != hipSuccess){ printf(" -hipMalloc d_covImgBin: %s\n",hipGetErrorString(err)); return 0;}

    err = hipMalloc((void**)&d_steImgRGB, imgSize);
    if(err != hipSuccess){ printf(" -hipMalloc d_steImgRGB: %s\n",hipGetErrorString(err)); return 0;}

    err = hipMalloc((void**)&d_steImgBin, imgSizeBin);
    if(err != hipSuccess){ printf(" -hipMalloc d_steImgBin: %s\n",hipGetErrorString(err)); return 0;}

    h_secImgRGB = secretImg.data;
    h_covImgRGB = coverImg.data;

    err = hipMemcpy(d_secImgRGB, h_secImgRGB, imgSize, hipMemcpyHostToDevice);
    if(err != hipSuccess){ printf(" -hipMemcpy d_secImgRGB < h_secImgRGB: %s\n",hipGetErrorString(err)); return 0;}

    err = hipMemcpy(d_covImgRGB, h_covImgRGB, imgSize, hipMemcpyHostToDevice);
    if(err != hipSuccess){ printf(" -hipMemcpy d_covImgRGB < h_covImgRGB: %s\n",hipGetErrorString(err)); return 0;}

    int threads = 32;
    dim3 blockDim(threads,threads);
	dim3 gridDim(ceil((float)rows/blockDim.x), ceil((float)rows/blockDim.y));

    // imgToBinGPU<<<gridDim, blockDim>>>(d_secImgRGB, d_secImgBin, colsRGB, rows);
    // err = hipDeviceSynchronize();
    // if(err != hipSuccess){ printf(" -Kernel call imgToBin(secImg): %s\n",hipGetErrorString(err)); return 0;}
    
    // err = hipMemcpy(h_secImgBin, d_secImgBin, imgSizeBin, hipMemcpyDeviceToHost);
    // if(err != hipSuccess){ printf(" -hipMemcpy h_secImgBin < d_secImgBin: %s\n",hipGetErrorString(err)); return 0;}

    // imgToBinGPU<<<gridDim, blockDim>>>(d_covImgRGB, d_covImgBin, colsRGB, rows);
    // err = hipDeviceSynchronize();
    // if(err != hipSuccess){ printf(" -Kernel call imgToBin(covImg): %s\n",hipGetErrorString(err)); return 0;}
    
    // err = hipMemcpy(h_covImgBin, d_covImgBin, imgSizeBin, hipMemcpyDeviceToHost);
    // if(err != hipSuccess){ printf(" -hipMemcpy h_covImgBin < d_covImgBin: %s\n",hipGetErrorString(err)); return 0;}
    
    clock_t startCPU = clock();

    imgToBin(h_secImgRGB, h_secImgBin, colsRGB, rows);
    imgToBin(h_covImgRGB, h_covImgBin, colsRGB, rows);
    hideImage(h_secImgBin, h_covImgBin, h_steImgBin, colsRGB, rows);
    imgToDec(h_steImgBin, h_steImgRGB, colsRGB, rows);
    getSecImg(h_steImgBin, h_secImgBin, colsRGB, rows);
    imgToDec(h_secImgBin, h_secImgRGB, colsRGB, rows);

    timeCPU = ((double)(clock() - startCPU))/CLOCKS_PER_SEC;

    printf("%f\n",timeCPU);

    stegoImg.create(rows, cols, CV_8UC3);
    stegoImg.data = h_steImgRGB;

    recovImg.create(rows, cols, CV_8UC3);
    recovImg.data = h_secImgRGB;

    imwrite("stegoImgOut.jpg", stegoImg);
    imwrite("secretImgRec.jpg", recovImg);

    return 0;
}