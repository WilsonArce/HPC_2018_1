#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

__global__ void gpuGrayScale(unsigned char *imgIn, unsigned char *imgOut, int cols, int rows){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned char r,g,b;
  if((row < rows) && (col < cols)){
    r = imgIn[(row * cols + col) * 3 + 2];
    g = imgIn[(row * cols + col) * 3 + 1];
    b = imgIn[(row * cols + col) * 3 + 0];

    imgOut[row * cols + col] = r * 0.299 + g * 0.587 + b * 0.114;
  }
}

__global__ void gpuSobelFilter(unsigned char *imgGray, unsigned char *imgFiltered, \
  unsigned char *imgX, unsigned char *imgY, int cols, int rows){
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.y + threadIdx.x;

  int xFilter[9] = {-1,0,1,-2,0,2,-1,0,1};
	int yFilter[9] = {-1,-2,-1,0,0,0,1,2,1};

  int sbCols, sbRows, sumx, sumy, x, y, ci, cj;
  sbCols = sbRows = 3;

  //for(i = 0; i < rows; i++){
	//	for(j = 0; j < cols; j++){
    if((i < rows) && (j < cols)){
			sumx = 0; sumy = 0; ci = i-2;
			for(y = 0; y < sbRows; y++){
				ci++;
				cj = j-1;
				for(x = 0; x < sbCols; x++){
					if(ci < 0 || cj < 0){
						sumx += 0;
						sumy += 0;
					}else{
						sumx += imgGray[ci * cols + cj] * xFilter[y * sbCols + x];
						sumy += imgGray[ci * cols + cj] * yFilter[y * sbCols + x];
					}
					cj++;
				}
			}
			if(sumx > 255){
				imgX[i * cols + j] = 255;
			}else{
				if(sumx < 0){
					imgX[i * cols + j] = 0;
				}else{
					imgX[i * cols + j] = sumx;
				}
			}
			if(sumy > 255){
				imgY[i * cols + j] = 255;
			}else{
				if(sumy < 0){
					imgY[i * cols + j] = 0;
				}else{
					imgY[i * cols + j] = sumy;
				}
			}
			imgFiltered[i * cols + j] = sqrt(powf(imgX[i * cols + j],2) + powf(imgY[i * cols + j],2));
		}
	//}

}


int main(int argc, char** argv )
{

  double timeGPU_GS, timeGPU_SB, timeCPU_GS, timeCPU_SB;

  //elements for GRAYSCALE filter
  unsigned char *h_imageIn, *h_imageGray, *d_imageIn, *d_imageGray;

  //elements for SOBEL filter
  unsigned char *h_imageSobel, *d_imageX, *d_imageY, *d_imageSobel;

  //char* window_name = "Sobel Demo - Simple Edge Detector";
  int scale = 1;
  int delta = 0;
  int ddepth = CV_16S;

  //hipError_t error = hipSuccess;
  Mat image;
  image = imread( argv[1], 1 );
  
  if ( argc != 2 )
  {
    printf("usage: DisplayImage <Image_Path>\n");
    return -1;
  }

  //+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
  ///Element for openCV transformations
  Mat src_gray;
  Mat grad;

  clock_t startCPU_GS = clock();
  /// Convert it to gray
  cvtColor( image, src_gray, CV_BGR2GRAY );
  timeCPU_GS = ((double)(clock() - startCPU_GS))/CLOCKS_PER_SEC;

  /// Generate grad_x and grad_y
  Mat grad_x, grad_y;
  Mat abs_grad_x, abs_grad_y;

  clock_t startCPU_SB = clock();
  /// Gradient X
  //Scharr( src_gray, grad_x, ddepth, 1, 0, scale, delta, BORDER_DEFAULT );
  Sobel( src_gray, grad_x, ddepth, 1, 0, 3, scale, delta, BORDER_DEFAULT );
  convertScaleAbs( grad_x, abs_grad_x );

  /// Gradient Y
  //Scharr( src_gray, grad_y, ddepth, 0, 1, scale, delta, BORDER_DEFAULT );
  Sobel( src_gray, grad_y, ddepth, 0, 1, 3, scale, delta, BORDER_DEFAULT );
  convertScaleAbs( grad_y, abs_grad_y );

  /// Total Gradient (approximate)
  addWeighted( abs_grad_x, 0.5, abs_grad_y, 0.5, 0, grad );
  timeCPU_SB = ((double)(clock() - startCPU_SB))/CLOCKS_PER_SEC;
  //+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

  //image size cols and rows
  int cols = image.cols;
  int rows = image.rows;

  //size of initial image and result images
  int imgInSize = sizeof(unsigned char) * cols * rows * image.channels();
  int imgOutSize = sizeof(unsigned char) * cols * rows;

  //allocation of memory for elements of GRAYSCALE filter ON HOST
  h_imageIn = (unsigned char*)malloc(imgInSize);
  h_imageGray = (unsigned char*)malloc(imgOutSize);

  //allocation of memory for elements of GRAYSCALE filter ON DEVICE
  hipMalloc((void**)&d_imageIn, imgInSize);
  hipMalloc((void**)&d_imageGray, imgOutSize);

  //allocation of memory for elements of SOBEL filter ON HOST
  h_imageSobel = (unsigned char*)malloc(imgOutSize);

  //allocation of memory for elements of SOBEL filter ON DEVICE
  hipMalloc((void**)&d_imageX, imgOutSize);
  hipMalloc((void**)&d_imageY, imgOutSize);
  hipMalloc((void**)&d_imageSobel, imgOutSize);

  //error = hipMalloc((void**)&d_imageIn, imgInSize);
  /*if(error != hipSuccess){
      printf("Error reservando memoria para d_imageIn\n -> %s\n", hipGetErrorString(error));
      exit(-1);
  }*/
  
  //passing data for image processing
  h_imageIn = image.data;

  //passing image data from HOST to DEVICE for GRAYSCALE filter
  hipMemcpy(d_imageIn, h_imageIn, imgInSize, hipMemcpyHostToDevice);

  //parameters definition for CUDA kernel
  int threads = 32;
  dim3 numThreads(threads, threads);
  dim3 blockDim(ceil(cols/float(threads)), ceil(rows/float(threads)));

  clock_t startGPU_GS = clock();
  //CUDA grayscale kernel call
  gpuGrayScale<<<blockDim, numThreads>>>(d_imageIn, d_imageGray, cols, rows);
  hipDeviceSynchronize();//CUDA threads sincronization
  timeGPU_GS = ((double)(clock() - startGPU_GS))/CLOCKS_PER_SEC;

  //passing result GRAYSCALE data from DEVICE to HOST
  hipMemcpy(h_imageGray, d_imageGray, imgOutSize, hipMemcpyDeviceToHost);

  clock_t startGPU_SB = clock();
  //CUDA sobel filter call
  gpuSobelFilter<<<blockDim, numThreads>>>(d_imageGray, d_imageSobel, d_imageX, d_imageY, cols, rows);
  hipDeviceSynchronize();//CUDA threads sincronization
  timeGPU_SB = ((double)(clock() - startGPU_SB))/CLOCKS_PER_SEC;

  //passing result SOBEL data from DEVICE to HOST
  hipMemcpy(h_imageSobel, d_imageSobel, imgOutSize, hipMemcpyDeviceToHost);

  Mat imageOut;
  imageOut.create(rows, cols, CV_8UC1);
  imageOut.data = h_imageSobel;

  //printf("**Global memory implementation**\n");
  //cout<<"Image size = "<< image.size() << endl;
  //printf("Global memory\nGrayscale time\n");
  printf("%f",timeGPU_SB);
  //printf("Sobel filter time\n");
  //printf("sf{%f-%f}",timeCPU_SB,timeGPU_SB);
  //printf("  CPU = %f s\n",timeCPU_SB);
  //printf("  GPU = %f s\n",timeGPU_SB);

  imwrite("imageSobel_gpu.jpg", imageOut);
  imwrite("imageSobel_opCV.jpg", grad);

  //waitKey(0);

  //memory deallocation on DEVICE
  hipFree(d_imageIn);
  hipFree(d_imageGray);
  hipFree(d_imageX);
  hipFree(d_imageY);

  return 0;
}