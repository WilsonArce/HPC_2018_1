#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

__global__ void gpuGrayScale(unsigned char *imgIn, unsigned char *imgOut, int cols, int rows){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned char r,g,b;
  if((row < rows) && (col < cols)){
    r = imgIn[(row * cols + col) * 3 + 2];
    g = imgIn[(row * cols + col) * 3 + 1];
    b = imgIn[(row * cols + col) * 3 + 0];

    imgOut[row * cols + col] = r * 0.299 + g * 0.587 + b * 0.114;
  }
}


int main(int argc, char** argv )
{

  unsigned char *imageIn, *h_imageOut, *d_imageIn, *d_imageOut;
  //hipError_t error = hipSuccess;
  Mat image;
  image = imread( argv[1], 1 );
  
  if ( argc != 2 )
  {
    printf("usage: DisplayImage <Image_Path>\n");
    return -1;
  }

  int cols = image.cols;
  int rows = image.rows;

  int imgInSize = sizeof(unsigned char) * cols * rows * image.channels();
  int imgOutSize = sizeof(unsigned char) * cols * rows;

  imageIn = (unsigned char*)malloc(imgInSize);
  h_imageOut = (unsigned char*)malloc(imgOutSize);

  hipMalloc((void**)&d_imageIn, imgInSize);

  //error = hipMalloc((void**)&d_imageIn, imgInSize);
  /*if(error != hipSuccess){
      printf("Error reservando memoria para d_imageIn\n -> %s\n", hipGetErrorString(error));
      exit(-1);
  }*/
  hipMalloc((void**)&d_imageOut, imgOutSize);

  imageIn = image.data;

  hipMemcpy(d_imageIn, imageIn, imgInSize, hipMemcpyHostToDevice);

  int threads = 32;
  dim3 numThreads(threads, threads);
  dim3 blockDim(ceil(cols/float(threads)), ceil(rows/float(threads)));

  gpuGrayScale<<<blockDim, numThreads>>>(d_imageIn, d_imageOut, cols, rows);
  hipDeviceSynchronize();

  hipMemcpy(h_imageOut, d_imageOut, imgOutSize, hipMemcpyDeviceToHost);

  Mat imageOut;
  imageOut.create(rows, cols, CV_8UC1);
  imageOut.data = h_imageOut;

  cout << imageOut.channels() << endl << sizeof(d_imageOut) << endl;

  imwrite("lena_out.jpg", imageOut);

  //waitKey(0);

  hipFree(d_imageIn);
  hipFree(d_imageOut);

  return 0;
}